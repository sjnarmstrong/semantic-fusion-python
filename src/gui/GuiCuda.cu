/*
 * This file is part of SemanticFusion.
 *
 * Copyright (C) 2017 Imperial College London
 * 
 * The use of the code within this file and all code within files that 
 * make up the software that is SemanticFusion is permitted for 
 * non-commercial purposes only.  The full terms and conditions that 
 * apply to the code within this file are detailed within the LICENSE.txt 
 * file and at <http://www.imperial.ac.uk/dyson-robotics-lab/downloads/semantic-fusion/semantic-fusion-license/> 
 * unless explicitly stated.  By downloading this file you agree to 
 * comply with these terms.
 *
 * If you wish to use any of this code for commercial purposes then 
 * please email researchcontracts.engineering@imperial.ac.uk.
 *
 */

#include <stdio.h>
#include <assert.h> 

#include <hip/hip_runtime.h>

#define gpuErrChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool
        abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n",
                hipGetErrorString(code), file, line);
        if (abort) exit(code);
    } 
}

__global__ 
void colouredArgMaxKernel(int n, float const* probabilities,  const int num_classes, float const* color_lookup, float* colour, float const* map_max, const int map_size,hipTextureObject_t ids, const float threshold)
{
    const int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n) {
        const int y = id / 640;
        const int x = id - (y * 640);
        const int start_windowx = (x - 1) > 0 ? (x - 1) : 0;
        const int start_windowy = (y - 1) > 0 ? (y - 1) : 0;
        const int end_windowx = (x + 1) < 640 ? (x + 1) : 639;
        const int end_windowy = (y + 1) < 480 ? (y + 1) : 479;

        int max_class_id = -1;
        float max_class_prob = threshold;
        for (int sx = start_windowx; sx <= end_windowx; ++sx) {
            for (int sy = start_windowy; sy <= end_windowy; ++sy) {
                const int surfel_id = tex2D<int>(ids,sx,sy);
                if (surfel_id > 0) {
                    float const* id_probabilities = map_max + surfel_id;
                    if (id_probabilities[map_size] > max_class_prob) {
                        max_class_id = static_cast<int>(id_probabilities[0]);
                        max_class_prob = id_probabilities[map_size];
                    }
                }
            }
        }

        float* local_colour = colour + (id * 4);
        if (max_class_id >= 0) {
            local_colour[0] = color_lookup[max_class_id * 3 + 0];
            local_colour[1] = color_lookup[max_class_id * 3 + 1];
            local_colour[2] = color_lookup[max_class_id * 3 + 2];
            local_colour[3] = 1.0f;
        } else {
            local_colour[0] = 0.0;
            local_colour[1] = 0.0;
            local_colour[2] = 0.0;
            local_colour[3] = 1.0f;
        }
    }
}

__host__
void colouredArgMax(int n, float const * probabilities,  const int num_classes, float const* color_lookup, float* colour, float const * map, const int map_size,hipTextureObject_t ids, const float threshold)
{
    const int threads = 512;
    const int blocks = (n + threads - 1) / threads;
    dim3 dimGrid(blocks);
    dim3 dimBlock(threads);
    colouredArgMaxKernel<<<dimGrid,dimBlock>>>(n,probabilities,num_classes,color_lookup,colour,map,map_size,ids,threshold);
    gpuErrChk(hipGetLastError());
    gpuErrChk(hipDeviceSynchronize());
}
