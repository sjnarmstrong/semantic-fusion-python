/*
 * This file is part of SemanticFusion.
 *
 * Copyright (C) 2017 Imperial College London
 * 
 * The use of the code within this file and all code within files that 
 * make up the software that is SemanticFusion is permitted for 
 * non-commercial purposes only.  The full terms and conditions that 
 * apply to the code within this file are detailed within the LICENSE.txt 
 * file and at <http://www.imperial.ac.uk/dyson-robotics-lab/downloads/semantic-fusion/semantic-fusion-license/> 
 * unless explicitly stated.  By downloading this file you agree to 
 * comply with these terms.
 *
 * If you wish to use any of this code for commercial purposes then 
 * please email researchcontracts.engineering@imperial.ac.uk.
 *
 */

#include <stdio.h>
#include <assert.h> 

#include <hip/hip_runtime.h>

#define gpuErrChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool
        abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n",
                hipGetErrorString(code), file, line);
        if (abort) exit(code);
    } 
}

__global__ 
void semanticTableUpdate(hipTextureObject_t ids, const int ids_width, const int ids_height, 
                          const float* probabilities, const int prob_width, const int prob_height, 
                          const int prob_channels,float* map_table,float* map_max,
                          const int map_size)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    // New uniqueness code
    const int check_patch = 16;
    const int x_min = (x - check_patch) < 0 ? 0 : (x - check_patch);
    const int x_max = (x + check_patch) > 640 ? 640 : (x + check_patch);
    const int y_min = (y - check_patch) < 0 ? 0 : (y - check_patch);
    int surfel_id = tex2D<int>(ids,x,y);
    int first_h, first_w;
    for (int h = y_min; h < 480; ++h) {
        for (int w = x_min; w < x_max; ++w) {
            int other_surfel_id = tex2D<int>(ids,w,h);
            if (other_surfel_id == surfel_id) {
                first_h = h;
                first_w = w;
                break;
            }
        }
    }
    if (first_h != y || first_w != x) {
        surfel_id = 0;
    }
    if (surfel_id > 0) {
        const int prob_x = static_cast<int>((float(x) / ids_width) * prob_width);
        const int prob_y = static_cast<int>((float(y) / ids_height) * prob_height);
        const int channel_offset = prob_width * prob_height;
        const float* probability = probabilities + (prob_y * prob_width + prob_x);
        float* prior_probability = map_table + surfel_id;
        float total = 0.0;
        for (int class_id = 0; class_id < prob_channels; ++class_id) {
            prior_probability[0] *= probability[0];
            total += prior_probability[0];
            probability += channel_offset;
            prior_probability += map_size;
        }
        // Reset the pointers to the beginning again
        probability = probabilities + (prob_y * prob_width + prob_x);
        prior_probability = map_table + surfel_id;
        float max_probability = 0.0;
        int max_class = -1;
        float new_total = 0.0;
        for (int class_id = 0; class_id < prob_channels; ++class_id) {
            // Something has gone unexpectedly wrong - reinitialse
            if (total <= 1e-5) {
                prior_probability[0] = 1.0f / prob_channels;
            } else {
                prior_probability[0] /= total;
                if (class_id > 0 && prior_probability[0] > max_probability) {
                    max_probability = prior_probability[0];
                    max_class = class_id;
                }
            }
            new_total += prior_probability[0];
            probability += channel_offset;
            prior_probability += map_size;
        }
        map_max[surfel_id] = static_cast<float>(max_class);
        map_max[surfel_id + map_size] = max_probability;
        map_max[surfel_id + map_size + map_size] += 1.0;
    }
}

__host__ 
void fuseSemanticProbabilities(hipTextureObject_t ids, const int ids_width, const int ids_height, 
                          const float* probabilities, const int prob_width, const int prob_height, 
                          const int prob_channels,float* map_table, float* map_max,
                          const int map_size)
{
    // NOTE Res must be pow 2 and > 32
    const int blocks = 32;
    dim3 dimGrid(blocks,blocks);
    dim3 dimBlock(640/blocks,480/blocks);
    semanticTableUpdate<<<dimGrid,dimBlock>>>(ids,ids_width,ids_height,probabilities,prob_width,prob_height,prob_channels,map_table,map_max,map_size);
    gpuErrChk(hipGetLastError());
    gpuErrChk(hipDeviceSynchronize());
}

__global__ 
void updateTable(int n, const int* deleted_ids, const int num_deleted, const int current_table_size,
                 float const* probability_table, const int prob_width, const int prob_height, 
                 const int new_prob_width, float* new_probability_table, float const * map_table, float* new_map_table)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        const int class_id = index / new_prob_width;
        const int component_id = index - (class_id * new_prob_width);
        const int new_id = (class_id * prob_width) + component_id;
        if (component_id >= num_deleted) {
            // Initialise to prior (prob height is the number of classes)
            new_probability_table[new_id] = 1.0f / prob_height;
            // Reset the max class surfel colouring lookup
            new_map_table[component_id] = -1.0;
            new_map_table[component_id + prob_width] = -1.0;
            new_map_table[component_id + prob_width + prob_width] = 0.0;
        } else {
            int offset = deleted_ids[component_id];
            new_probability_table[new_id] = probability_table[(class_id * prob_width) + offset];
            // Also must update our max class mapping
            new_map_table[component_id] = map_table[offset];
            new_map_table[component_id + prob_width] = map_table[prob_width + offset];
            new_map_table[component_id + prob_width + prob_width] = map_table[prob_width + prob_width + offset];
        }
    }
}

__host__ 
void updateProbabilityTable(int* filtered_ids, const int num_filtered, const int current_table_size,
                            float const* probability_table, const int prob_width, const int prob_height, 
                            const int new_prob_width, float* new_probability_table, 
                            float const* map_table, float* new_map_table)
{
    const int threads = 512;
    const int num_to_update = new_prob_width * prob_height;
    const int blocks = (num_to_update + threads - 1) / threads;
    dim3 dimGrid(blocks);
    dim3 dimBlock(threads);
    updateTable<<<dimGrid,dimBlock>>>(num_to_update,filtered_ids,num_filtered,current_table_size,probability_table,prob_width,prob_height,new_prob_width,new_probability_table, map_table, new_map_table);
    gpuErrChk(hipGetLastError());
    gpuErrChk(hipDeviceSynchronize());
}


__global__ 
void renderProbabilityMapKernel(hipTextureObject_t ids, const int ids_width, const int ids_height, 
                          const float* probability_table, const int prob_width, const int prob_height, 
                          float* rendered_probabilities) 
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int surfel_id = tex2D<int>(ids,x,y);
    int projected_probability_offset = y * ids_width + x;
    int probability_table_offset = surfel_id;
    for (int class_id = 0; class_id < prob_height; ++class_id) {
        if (surfel_id > 0) {
            rendered_probabilities[projected_probability_offset] = probability_table[probability_table_offset];
        } else {
            rendered_probabilities[projected_probability_offset] = ((class_id == 0) ? 1.0 : 0.0);
        }
        projected_probability_offset += (ids_width * ids_height);
        probability_table_offset += prob_width;
    }
}


__host__
void renderProbabilityMap(hipTextureObject_t ids, const int ids_width, const int ids_height, 
                          const float* probability_table, const int prob_width, const int prob_height, 
                          float* rendered_probabilities) 
{
    // NOTE Res must be pow 2 and > 32
    const int blocks = 32;
    dim3 dimGrid(blocks,blocks);
    dim3 dimBlock(ids_width/blocks,ids_height/blocks);
    renderProbabilityMapKernel<<<dimGrid,dimBlock>>>(ids,ids_width,ids_height,probability_table,prob_width,prob_height,rendered_probabilities);
    gpuErrChk(hipGetLastError());
    gpuErrChk(hipDeviceSynchronize());
}

__global__ 
void updateMaxClassKernel(const int n, const float* probabilities, const int classes,
                          float* map_max, const int map_size)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        // Reset the pointers to the beginning again
        const float* probability = probabilities + index;
        probability += map_size;
        float max_probability = 0.0;
        int max_class = -1;
        for (int class_id = 1; class_id < classes; ++class_id) {
            if (probability[0] > max_probability) {
                max_probability = probability[0];
                max_class = class_id;
            }
            probability += map_size;
        }
        map_max[index] = static_cast<float>(max_class);
        map_max[index + map_size] = max_probability;
    }
}

__host__ 
void updateMaxClass(const int n, const float* probabilities, const int classes,
                    float* map_max, const int map_size)
{
    const int threads = 512;
    const int blocks = (n + threads - 1) / threads;
    dim3 dimGrid(blocks);
    dim3 dimBlock(threads);
    updateMaxClassKernel<<<dimGrid,dimBlock>>>(n,probabilities,classes,map_max,map_size);
    gpuErrChk(hipGetLastError());
    gpuErrChk(hipDeviceSynchronize());
}
